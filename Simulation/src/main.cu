#include "hip/hip_runtime.h"
#include <exception>
#include <iostream>
#include "particles.h"
#include "distributor.h"

int main(int argc, char *argv[]) {
  try {
    // Probably should have some kind of builder
    Distributor <float,three_dimensional> distributor;

    auto parameters  = new Parameters<float, three_dimensional>;

    distributor.sync_from_renderer(*parameters);

    // @todo enforce dynamic construction requirment
    // The use of new isn't enforced but is required as managed memory is used
    auto particles = new Particles<float, three_dimensional>(*parameters);

    // After particles have been created construct initial fluid
    distributor.initilize_fluid(*particles, *parameters);

    // Sync initial particle configuration
    distributor.sync_to_renderer(*particles);

    int64_t frame = 0;

    // Number of simulation frames per render frame
    const int target_fps = 60;
    const int frames_per_update = (int)std::round(1.0 / parameters->time_step() / target_fps);
    std::cout<<"Compute updating renderer every "<<frames_per_update<<" frames"<<std::endl;

    // Main time step loop
    while(parameters->simulation_active()) {
      if(frame % frames_per_update == 0)
        distributor.sync_from_renderer(*parameters);

      if(parameters->compute_active()) {
        distributor.process_parameters(*parameters, *particles);

        particles->apply_external_forces(distributor.resident_span());

        particles->predict_positions(distributor.resident_span());

        distributor.balance_domains();

        distributor.domain_sync(*particles);

        particles->find_neighbors(distributor.local_span(),
                                 distributor.resident_span());

        for(int sub=0; sub<parameters->solve_step_count(); sub++) {
          particles->compute_densities(distributor.resident_span());

          particles->compute_pressure_lambdas(distributor.resident_span());
          distributor.initiate_sync_halo_scalar(particles->lambdas());
          distributor.finalize_sync_halo_scalar();

          particles->compute_pressure_dps(distributor.resident_span(), sub);

          particles->update_position_stars(distributor.resident_span());
          distributor.initiate_sync_halo_vec(particles->position_stars());
          distributor.finalize_sync_halo_vec();

          //        particles_.compute_surface_lambdas(distributor_.local_span());
          //        particles_.compute_surface_dps(distributor_.local_span(), sub);
        }

        particles->update_velocities(distributor.local_span());

        distributor.initiate_sync_halo_scalar(particles->densities());
        distributor.finalize_sync_halo_scalar();

        particles->apply_surface_tension(distributor.local_span(), distributor.resident_span());

        particles->apply_viscosity(distributor.resident_span());

        distributor.initiate_sync_halo_vec(particles->velocities());
        distributor.finalize_sync_halo_vec();

        particles->compute_vorticity(distributor.resident_span());

        distributor.initiate_sync_halo_vec(particles->scratch());
        distributor.finalize_sync_halo_vec();

        particles->apply_vorticity(distributor.resident_span());

        particles->update_positions(distributor.resident_span());

        // Needs to be done once per rendered frame
        if(frame % frames_per_update == 0)
          distributor.sync_to_renderer(*particles);

        frame++;

        distributor.invalidate_halo(*particles);
      }

    }

    delete parameters;
    delete particles;

  } catch(std::exception const& exception) {
      std::cout << "Aborting: " << exception.what() << std::endl;
      return 1;
  } catch(...) {
    std::cout << "Aborting: unknown exception" <<std::endl;
    return 1;
  }

  return 0;
}
